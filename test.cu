/*
   *mariの
   *curandSetPseudoRandomGeneratorSeed(mari, (unsigned long)time(NULL));
   *が反映されていない？
   *同じ値が帰ってくる
   */
#include<stdio.h>
#include<time.h>
#include<hiprand/hiprand.h>
#include<hip/hip_runtime.h>

int main(void) {
	float *d_array_rei;
	float *d_array_mari;
	float *h_array_rei;
	float *h_array_mari;
	size_t n = 16384;
	unsigned int dimension = 2;
	float mean = 0.0;
	float standard_deviation = 1.0;
	int i;
	//これが乱数生成器を指す名前的なもの
	//二つ作ってみる
	hiprandGenerator_t rei, mari;
	
	//デバイスとホストにメモリを確保する
	hipMalloc((void **)&d_array_rei, n * sizeof(float));
	hipMalloc((void **)&d_array_mari, n * sizeof(float));
	hipHostAlloc((void **)&h_array_rei, n * sizeof(float), hipHostMallocMapped);
	hipHostAlloc((void **)&h_array_mari, n * sizeof(float), hipHostMallocMapped);

	//乱数生成器を作る
	//\->reiはXORWOWというアルゴリズムを使い擬似乱数を作る乱数生成器とする
	hiprandCreateGenerator(&rei, HIPRAND_RNG_PSEUDO_XORWOW);
	//\->mariはSOBOLというアルゴリズムを使い準乱数を作る乱数生成器とする
	hiprandCreateGenerator(&mari, HIPRAND_RNG_QUASI_SOBOL32);

	//乱数生成器にシードを与える。ULLは型、64bit符号なし整数
	hiprandSetPseudoRandomGeneratorSeed(rei, 890106ULL);
	hiprandSetPseudoRandomGeneratorSeed(mari, 890106ULL);
	//\->time()を使うならこちら
	hiprandSetPseudoRandomGeneratorSeed(rei, (unsigned long)time(NULL));
	hiprandSetPseudoRandomGeneratorSeed(mari, (unsigned long)time(NULL));

	//オフセットを伝える。これも64bit符号なし整数で指定する
	//\->reiにだけオフセットを設け、mariはオフセットなしとする
	hiprandSetGeneratorOffset(rei, 5ULL);

	//rei,mariに格納順を伝える、どちらもデフォルトでよかろう
	hiprandSetGeneratorOrdering(rei, HIPRAND_ORDERING_PSEUDO_DEFAULT);
	hiprandSetGeneratorOrdering(mari, HIPRAND_ORDERING_QUASI_DEFAULT);

	//準乱数については、何次元空間で均一に分布するかを指定できる
	hiprandSetQuasiRandomGeneratorDimensions(mari, dimension);

	//n個だけ乱数を作らせ、結果をd_arrayに収める
	//\->reiにはfloatの一様乱数をつくらせる
	hiprandGenerateUniform(rei, d_array_rei, n);
	//\->mariにはfloatの正規分布乱数をつくらせる
	hiprandGenerateNormal(mari, d_array_mari, n, mean, standard_deviation);

	//デバイスからホストへ生成された乱数を持ってくる
	hipMemcpy(h_array_rei, d_array_rei, n * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_array_mari, d_array_mari, n * sizeof(float), hipMemcpyDeviceToHost);

	printf("rei's result\n");
	for(i = 0; i < 10; i += 1) {
		printf("%d:%f\n", i, h_array_rei[i]);
	}
	printf("mari's result\n");
	for(i = 0; i < 10; i += 1) {
		printf("%d:%f\n", i, h_array_mari[i]);
	}

	//乱数生成器を消す
	hiprandDestroyGenerator(rei);
	hiprandDestroyGenerator(mari);
	//デバイスとホストのメモリを解放する
	hipFree(d_array_rei);
	hipFree(d_array_mari);
	hipHostFree(h_array_rei);
	hipHostFree(h_array_mari);
	return 0;
}
