#include "hip/hip_runtime.h"
#include<stdio.h>
#include<time.h>
#include"mt.h"

#define NUM_THREAD 1024
__device__ __constant__ int d_Np;

__global__ void reduce_array_shared_memory(int *array, int *array_reduced, int dim_array, int num_block) {
	__shared__ int array_shared[NUM_THREAD];
	int global_id = threadIdx.x + blockIdx.x * blockDim.x;
	int block_id = blockIdx.x;
	int local_id = threadIdx.x;
	int i, j;
	int iterate_max = 1 + dim_array / (NUM_THREAD * num_block);
	int iterate;

	for(iterate = 0; iterate < iterate_max; iterate += 1) {
		i = global_id + iterate * num_block * NUM_THREAD;
		if(i < d_Np) {
			array_shared[local_id] = array[i];
		} else {
			array_shared[local_id] = 0;
		}
		__syncthreads();

		for(j = NUM_THREAD / 2; j > 0; j /= 2) {
			if((local_id < j) && (local_id + j < dim_array)) {
				array_shared[local_id] += array_shared[local_id + j]; 
			}
		__syncthreads();
		}

		if(local_id == 0) {
			array_reduced[block_id] = array_shared[0];
		}
		__syncthreads();
		block_id += num_block;
	}
}

void fill_array(int *array, int dim) {
	int i;
	for(i = 0; i < dim; i += 1) {
		if(genrand_real1() < 0.5) {
			array[i] = 0;
		} else {
			array[i] = 1;
		}
	}
}

int h_count_active_particle(int *h_active, int h_Np) {
	int i;
	int sum;
	sum = 0;
	for(i = 0; i < h_Np; i += 1) {
		sum += h_active[i];
	}
	return sum;
}

int count_active_on_device(int *d_active, int h_Np, int num_block) {
	int i, j, k;
	int i_temp;
	int *d_reduction[2];
	int h_answer;
	hipMalloc((void **)&d_reduction[0], h_Np * sizeof(int));
	hipMalloc((void **)&d_reduction[1], h_Np * sizeof(int));
	i = 0;
	j = 1;
	hipMemcpy(d_reduction[i], d_active, h_Np * sizeof(int), hipMemcpyDeviceToDevice);
	for(k = h_Np; k > 1; k = 1 + k / NUM_THREAD) {
		reduce_array_shared_memory<<<num_block, NUM_THREAD>>>(d_reduction[i], d_reduction[j], k, num_block);
		hipDeviceSynchronize();
		i_temp = i;
		i = j;
		j = i_temp;
	}
	hipMemcpy(&h_answer, d_reduction[i], sizeof(int), hipMemcpyDeviceToHost);
	printf("gpu:%d, ", h_answer);

	hipFree(d_reduction[0]);
	hipFree(d_reduction[1]);
	return 0;
}

int main(void){
	int *h_active;
	int *d_active;
	int h_Np = 70000000;
	int h_ans;
	int num_block = 5;
	clock_t start, end;
	hipMalloc((void **)&d_active, h_Np * sizeof(int));
	hipHostAlloc((void **)&h_active, h_Np * sizeof(int), hipHostMallocMapped);
	hipMemcpyToSymbol(HIP_SYMBOL(d_Np), &h_Np, sizeof(int), 0, hipMemcpyHostToDevice);
	init_genrand((int)time(NULL));
	//init_genrand(19970303);

	fill_array(h_active, h_Np);
	start = clock();
	h_ans = h_count_active_particle(h_active, h_Np);
	end = clock();
	printf("cpu:%d, time:%d\n", h_ans, (int)(end - start));

	start = clock();
	hipMemcpy(d_active, h_active, h_Np * sizeof(int), hipMemcpyHostToDevice);
	count_active_on_device(d_active, h_Np, num_block);
	end = clock();
	printf("time:%d\n", (int)(end - start));

	hipFree(d_active);
	hipHostFree(h_active);

	return 0;
}
