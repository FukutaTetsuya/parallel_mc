#include "hip/hip_runtime.h"
/*
 * Cell(i, j) = cell[i + j * n]


 */
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<string.h>
#include<time.h>
#include"mt.h"

__device__ __constant__ int d_Np;
__device__ __constant__ double d_L;
#define NUM_BLOCK 32
#define NUM_THREAD 32
#define PI 3.1415926535897932384626433

//host functions----------------------------------------------------------------
void init_configuration(double *h_x, double *h_y, double h_L, int h_Np) {
	int i;

	for(i = 0; i < h_Np; i += 1) {
		h_x[i] = h_L * genrand_real2();
		h_y[i] = h_L * genrand_real2();
	}
}

void h_check_active(double *h_x, double *h_y, double h_L, int h_Np, int *h_active) {
	int i, j;
	double dx, dy, dr_square;
	double diameter_square = 1.0;

	for(i = 0; i < h_Np; i += 1) {
		h_active[i] = 0;
	}
	for(i = 0; i < h_Np; i += 1) {
		for(j = 0; j < i; j += 1) {
			dx = h_x[i] - h_x[j];
			if(dx > 0.5 * h_L) {
				dx -= h_L;
			} else if(dx < -0.5 * h_L) {
				dx += h_L;
			}
			dy = h_y[i] - h_y[j];
			if(dy > 0.5 * h_L) {
				dy -= h_L;
			} else if(dy < -0.5 * h_L) {
				dy += h_L;
			}

			dr_square = dx * dx + dy * dy;
			if(dr_square < diameter_square) {
				h_active[i] = 1;
				h_active[j] = 1;
			}
		}
	}
}

void h_check_active_with_list(double *h_x, double *h_y, double h_L, int h_Np, int *h_active, int *h_cell_list, int cell_per_axis, int N_per_cell) {
	int i, j;
	int x_c, y_c;
	int cell_id, N_in_cell;
	int pair_id;
	double dx, dy, dr_square;
	double diameter_square = 1.0;

	for(i = 0; i < h_Np; i += 1) {
		x_c = (int)(h_x[i] * (double)cell_per_axis / h_L);
		y_c = (int)(h_y[i] * (double)cell_per_axis / h_L);
		cell_id = x_c + y_c * cell_per_axis;
		N_in_cell = h_cell_list[cell_id * N_per_cell];
		for(j = 1; j <= N_in_cell; j += 1) {
			pair_id = h_cell_list[cell_id * N_per_cell + j];
			if(i == pair_id) {continue;}
			dx = h_x[i] - h_x[pair_id];
			if(dx < -0.5 * h_L) {
				dx += h_L;
			} else if(dx > 0.5 * h_L) {
				dx -= h_L;
			}
			dy = h_y[i] - h_y[pair_id];
			if(dy < -0.5 * h_L) {
				dy += h_L;
			} else if(dy > 0.5 * h_L) {
				dy -= h_L;
			}
			dr_square = dx * dx + dy * dy;
			if(diameter_square > dr_square) {
				h_active[i] = 1;
			}
		}
	}
}


void h_DBG(int *A, int *B, int dim) {
	int i;
	double res = 0.0;
	for(i = 0; i < dim; i += 1) {
		res += (A[i] - B[i]) * (A[i] - B[i]);
	}
	printf("res %f\n", res);
}

int h_make_cell_list(double *h_x, double *h_y, double h_L, int h_Np, int *h_cell_list, int cell_per_axis, int N_per_cell) {
	//I dont know which is better modulo (%)calculation and if(){}elseif(){}else{}
	int i, j, k;
//	int j_next, k_next;
	int x_cell, y_cell;
	int cell_id;
	int cell_list_size = cell_per_axis * cell_per_axis * N_per_cell;
	int contained_num;
	//init cell list
	for(i = 0; i < cell_list_size; i += 1) {
		h_cell_list[i] = 0;
	}
	//make cell list
	for(i = 0; i < h_Np; i += 1) {
		x_cell = (int)(h_x[i] * (double)cell_per_axis / h_L);
		y_cell = (int)(h_y[i] * (double)cell_per_axis / h_L);
		for(j = x_cell - 1; j <= x_cell + 1; j += 1) {
/*			if(j < 0) {
 *				j_next = j + cell_per_axis;
 *			} else if(j >= cell_per_axis) {
 *				j_next = j - cell_per_axis;
 *			} else {
 *				j_next = j;
 *			}
 */
			for(k = y_cell - 1; k <= y_cell + 1; k += 1) {
/*				if(k < 0) {
 *					k_next = k + cell_per_axis;
 *				} else if(k >= cell_per_axis) {
 *					k_next = k - cell_per_axis;
 *				} else {
 *					k_next = k;
 *				}
 */
				cell_id = ((j + cell_per_axis) % cell_per_axis) + ((k + cell_per_axis) % cell_per_axis) * cell_per_axis;
				//cell_id = j_next + k_next * cell_per_axis;
				h_cell_list[cell_id * N_per_cell] += 1;
				contained_num = h_cell_list[cell_id * N_per_cell];
				if(contained_num >= N_per_cell) {
					printf("too many particles in a cell\n");
					return 1;
				}
				h_cell_list[cell_id * N_per_cell + contained_num] = i;
			}
		}
	}
	return 0;
}


//device functions--------------------------------------------------------------
__global__ void d_check_active(double *d_x, double *d_y, int *d_active) {
	int i_global;
	int i, j;
	int Np = d_Np;
	double l = 0.5 * d_L;
	double dx, dy ,dr_square;
	double diameter_square = 1.0;

	i_global = blockDim.x * blockIdx.x + threadIdx.x;
	for(i = i_global; i < Np; i += NUM_BLOCK * NUM_THREAD) {
		d_active[i] = 0;
		for(j = 0; j < Np; j += 1) {
			if(j != i) {
				dx = d_x[i] - d_x[j];
				if(dx > l) {
					dx -= d_L;
				} else if(dx < -l) {
					dx += d_L;
				}
				dy = d_y[i] - d_y[j];
				if(dy > l) {
					dy -= d_L;
				} else if(dy < -l) {
					dy += d_L;
				}
				dr_square = dx * dx + dy * dy;

				if(dr_square < diameter_square) {
					d_active[i] = 1;
					break;
				}
			}
		}

	}
}

__global__ void d_check_active_with_list(double *d_x, double *d_y, int *d_active, int *d_cell_list, int cell_per_axis, int N_per_cell) {
	//d_L and d_Np are already declared as __global__ const
	int i, j;
	int x_c, y_c;
	int cell_id, N_in_cell;
	int pair_id;
	int i_global;
	double dx, dy, dr_square;
	double diameter_square = 1.0;
	i_global = blockDim.x * blockIdx.x + threadIdx.x;
	for(i = i_global; i < d_Np; i += NUM_BLOCK * NUM_THREAD) {
		d_active[i] = 0;
		x_c = (int)(d_x[i] * (double)cell_per_axis / d_L);
		y_c = (int)(d_y[i] * (double)cell_per_axis / d_L);
		cell_id = x_c + y_c * cell_per_axis;
		N_in_cell = d_cell_list[cell_id * N_per_cell];	
		for(j = 1; j <= N_in_cell; j += 1) {
			pair_id = d_cell_list[cell_id * N_per_cell + j];
			if(i == pair_id) {continue;}
			dx = d_x[i] - d_x[pair_id];
			dy = d_y[i] - d_y[pair_id];
			if(dx < -0.5 * d_L) {
				dx += d_L;
			} else if(dx > 0.5 * d_L) {
				dx -= d_L;
			}
			if(dy < -0.5 * d_L) {
				dy += d_L;
			} else if(dy > 0.5 * d_L) {
				dy -= d_L;
			}
			dr_square = dx * dx + dy * dy;
			if(diameter_square > dr_square) {
				d_active[i] = 1;
			}
		}
	}
}

__global__ void d_check_belonging_cell(double *d_x, double *d_y, int *d_cell_list, int *d_belonging_cell, int cell_per_axis, int N_per_cell) {
	//d_L and d_Np are already declared as __global__ const
	int i;
	int i_global;
	int x_cell, y_cell;
	int cell_id;

	i_global = blockDim.x * blockIdx.x + threadIdx.x;
	for(i = i_global; i < d_Np; i += NUM_BLOCK * NUM_THREAD) {
		x_cell = (int)(d_x[i] * (double)cell_per_axis / d_L);
		y_cell = (int)(d_y[i] * (double)cell_per_axis / d_L);
		cell_id = x_cell + y_cell * cell_per_axis;
		d_belonging_cell[i] = cell_id;
	}
}

__global__ void d_make_cell_list_from_belonging_cell(double *d_x, double *d_y, int *d_cell_list, int *d_belonging_cell, int cell_per_axis, int N_per_cell) {
	//d_L and d_Np are already declared as __global__ const
	//modulos or if()elseif(), which is the faster?
	int i, j, k, l;
	int i_global;
	int cell_id;
	int x_cell, y_cell;
	int x_next, y_next;

	i_global = blockDim.x * blockIdx.x + threadIdx.x;
	for(i = i_global; i < cell_per_axis * cell_per_axis; i += NUM_BLOCK * NUM_THREAD) {
		d_cell_list[i * N_per_cell] = 0;
		x_cell = i % cell_per_axis;
		y_cell = i / cell_per_axis;
		for(j = x_cell - 1; j <= x_cell + 1; j += 1) {
/*			if(j < 0) {
				x_next = j + cell_per_axis;
			} else if(j >= cell_per_axis) {
					x_next = j - cell_per_axis;
			} else {
					x_next = j;
			}
 */
			for(k = y_cell - 1; k <= y_cell + 1; k += 1) {
/*				if(k < 0) {
					y_next = k + cell_per_axis;
				} else if(k >= cell_per_axis) {
					y_next = k - cell_per_axis;
				} else {
					y_next = k;
				}
 */
				cell_id = ((j + cell_per_axis) % cell_per_axis) + ((k + cell_per_axis) % cell_per_axis) * cell_per_axis;
				//cell_id = x_next + y_next * cell_per_axis;
				for(l = 0; l < d_Np; l += 1) {
					if(d_belonging_cell[l] == cell_id) {
						d_cell_list[i * N_per_cell] += 1;
						d_cell_list[i * N_per_cell +  d_cell_list[i * N_per_cell] ] = l;
					}
				}
			}
		}
	}

}

void h_make_cell_list_on_device(double *d_x, double *d_y, int *d_cell_list, int *d_belonging_cell, int cell_per_axis, int N_per_cell) {
	//d_L and d_Np are already declared as __global__ const

	//check belonging cell
	d_check_belonging_cell<<<NUM_BLOCK, NUM_THREAD>>>(d_x, d_y, d_cell_list, d_belonging_cell, cell_per_axis, N_per_cell);
	//synchronize
	hipDeviceSynchronize();
	//gather the belonging cell and make cell list
	d_make_cell_list_from_belonging_cell<<<NUM_BLOCK, NUM_THREAD>>>(d_x, d_y, d_cell_list, d_belonging_cell, cell_per_axis, N_per_cell);
	//synchronize
	hipDeviceSynchronize();
}

//------------------------------------------------------------------------------
int main(void) {
	int i;
	clock_t start, end;
	int cell_per_axis;
	int N_per_cell;
	FILE *file;

	//variables in host
	double *h_x;
	double *h_y;
	double h_L;
	int *h_active;
	int *h_check_result;
	int h_Np;
	int *h_cell_list;
	int *h_active_DBG;

	//variables in device
	double *d_x;
	double *d_y;
	int *d_active;
	int *d_cell_list;
	int *d_belonging_cell;

	//initialize
	//init_genrand(19970303);
	init_genrand((int)time(NULL));

	//--set variable
	h_Np = 18000;
	h_L = 140.0;
	cell_per_axis = (int)(h_L / 11.0) + 1;//renew list every 5 steps
	N_per_cell = (h_Np * 13) / (cell_per_axis * cell_per_axis);
	printf("cell per axis:%d N_per_cell:%d\n", cell_per_axis, N_per_cell);

	hipMemcpyToSymbol(HIP_SYMBOL(d_Np), &h_Np, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_L), &h_L, sizeof(double), 0, hipMemcpyHostToDevice);

	//--allocate memory
	//----memory on host
	hipHostAlloc((void **)&h_x, h_Np * sizeof(double), hipHostMallocMapped);
	hipHostAlloc((void **)&h_y, h_Np * sizeof(double), hipHostMallocMapped);
	hipHostAlloc((void **)&h_active, h_Np * sizeof(int), hipHostMallocMapped);
	hipHostAlloc((void **)&h_check_result, h_Np * sizeof(int), hipHostMallocMapped);
	hipHostAlloc((void **)&h_cell_list, cell_per_axis * cell_per_axis * N_per_cell * sizeof(int), hipHostMallocMapped);
	h_active_DBG = (int *)calloc(h_Np, sizeof(int));

	//----memory on device
	hipMalloc((void **)&d_x, h_Np * sizeof(double));
	hipMalloc((void **)&d_y, h_Np * sizeof(double));
	hipMalloc((void **)&d_active, h_Np * sizeof(int));
	hipMalloc((void **)&d_cell_list, cell_per_axis * cell_per_axis * N_per_cell * sizeof(int));
	hipMalloc((void **)&d_belonging_cell, h_Np * sizeof(int));

	//--place particles
	init_configuration(h_x, h_y, h_L, h_Np);
	hipMemcpy(d_x, h_x, h_Np * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, h_Np * sizeof(double), hipMemcpyHostToDevice);

	//--make first acriveness array
	//----made in host
	start = clock();
	h_check_active(h_x, h_y, h_L, h_Np, h_active);
	end = clock();
	printf("straighforward:%d [ms]\n\n", (int)((end - start)*1000 /CLOCKS_PER_SEC ));

	//----made in host with cell list
	start = clock();
	h_make_cell_list(h_x, h_y, h_L, h_Np, h_cell_list, cell_per_axis, N_per_cell);
	h_check_active_with_list(h_x, h_y, h_L, h_Np, h_active_DBG, h_cell_list, cell_per_axis, N_per_cell);
	end = clock();
	printf("host cell list:%d [ms]\n", (int)((end - start)*1000 /CLOCKS_PER_SEC ));
	h_DBG(h_active, h_active_DBG, h_Np);
	printf("\n");

	//----made in device global
	start = clock();
	hipMemcpy(d_x, h_x, h_Np * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, h_Np * sizeof(double), hipMemcpyHostToDevice);
	d_check_active<<<NUM_BLOCK, NUM_THREAD>>>(d_x, d_y, d_active);
	hipDeviceSynchronize();
	hipMemcpy(h_check_result, d_active, h_Np * sizeof(int), hipMemcpyDeviceToHost);
	end = clock();
	printf("gpu:%d [ms]\n", (int)((end - start)*1000 /CLOCKS_PER_SEC ));
	h_DBG(h_active, h_check_result, h_Np);
	printf("\n");

	//----made in device global with list, list is made in host
	start = clock();
	h_make_cell_list(h_x, h_y, h_L, h_Np, h_cell_list, cell_per_axis, N_per_cell);
	hipMemcpy(d_x, h_x, h_Np * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, h_Np * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_cell_list, h_cell_list, N_per_cell * cell_per_axis * cell_per_axis * sizeof(int), hipMemcpyHostToDevice);
	d_check_active_with_list<<<NUM_BLOCK, NUM_THREAD>>>(d_x, d_y, d_active, d_cell_list, cell_per_axis, N_per_cell);
	hipDeviceSynchronize();
	hipMemcpy(h_check_result, d_active, h_Np * sizeof(int), hipMemcpyDeviceToHost);
	end = clock();
	printf("gpu with host list:%d [ms]\n", (int)((end - start)*1000 /CLOCKS_PER_SEC ));
	h_DBG(h_active, h_check_result, h_Np);
	printf("\n");

	//----made in device global with list, list is made in device
	start = clock();
	hipMemcpy(d_x, h_x, h_Np * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, h_Np * sizeof(double), hipMemcpyHostToDevice);
	h_make_cell_list_on_device(d_x, d_y, d_cell_list, d_belonging_cell, cell_per_axis, N_per_cell);
	hipDeviceSynchronize();
	d_check_active_with_list<<<NUM_BLOCK, NUM_THREAD>>>(d_x, d_y, d_active, d_cell_list, cell_per_axis, N_per_cell);
	hipDeviceSynchronize();
	hipMemcpy(h_check_result, d_active, h_Np * sizeof(int), hipMemcpyDeviceToHost);
	end = clock();
	printf("gpu with gpu list:%d [ms]\n", (int)((end - start)*1000 /CLOCKS_PER_SEC ));
	h_DBG(h_active, h_check_result, h_Np);
	printf("\n");

	//time loop
	//--move particles
	//--check activeness
	//--(sometimes) make new cell list

	//finalize
	//--free memory
	hipHostFree(h_x);
	hipHostFree(h_y);
	hipHostFree(h_active);
	hipHostFree(h_check_result);
	hipHostFree(h_cell_list);
	free(h_active_DBG);

	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_active);
	hipFree(d_cell_list);
	hipFree(d_belonging_cell);
	return 0;
}
